#include "hip/hip_runtime.h"
/* -*- mode: C; tab-width: 2; indent-tabs-mode: nil; -*- */

/*
 * This code has been contributed by the DARPA HPCS program.  Contact
 * David Koester <dkoester@mitre.org> or Bob Lucas <rflucas@isi.edu>
 * if you have questions.
 *
 * GUPS (Giga UPdates per Second) is a measurement that profiles the memory
 * architecture of a system and is a measure of performance similar to MFLOPS.
 * The HPCS HPCchallenge RandomAccess benchmark is intended to exercise the
 * GUPS capability of a system, much like the LINPACK benchmark is intended to
 * exercise the MFLOPS capability of a computer.  In each case, we would
 * expect these benchmarks to achieve close to the "peak" capability of the
 * memory system. The extent of the similarities between RandomAccess and
 * LINPACK are limited to both benchmarks attempting to calculate a peak system
 * capability.
 *
 * GUPS is calculated by identifying the number of memory locations that can be
 * randomly updated in one second, divided by 1 billion (1e9). The term "randomly"
 * means that there is little relationship between one address to be updated and
 * the next, except that they occur in the space of one half the total system
 * memory.  An update is a read-modify-write operation on a table of 64-bit words.
 * An address is generated, the value at that address read from memory, modified
 * by an integer operation (add, and, or, xor) with a literal value, and that
 * new value is written back to memory.
 *
 * We are interested in knowing the GUPS performance of both entire systems and
 * system subcomponents --- e.g., the GUPS rating of a distributed memory
 * multiprocessor the GUPS rating of an SMP node, and the GUPS rating of a
 * single processor.  While there is typically a scaling of FLOPS with processor
 * count, a similar phenomenon may not always occur for GUPS.
 *
 * For additional information on the GUPS metric, the HPCchallenge RandomAccess
 * Benchmark,and the rules to run RandomAccess or modify it to optimize
 * performance -- see http://icl.cs.utk.edu/hpcc/
 *
 */

/*
 * This file contains the computational core of the single cpu version
 * of GUPS.  The inner loop should easily be vectorized by compilers
 * with such support.
 *
 * This core is used by both the single_cpu and star_single_cpu tests.
 */

#include <hpcc.h>
extern "C" {
int HPCC_MPIRandomAccess(HPCC_Params *params);
}
#include <sys/time.h>
#include <sys/resource.h>
#include "RandomAccess.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
/* Number of updates to table (suggested: 4x number of table entries) */
#define NUPDATE (4 * TableSize)

static __constant__ uint64_t c_m2[64];
static __device__ uint32_t d_error[1];

double CPUSEC_SAME()
{
   static double              cps = CLOCKS_PER_SEC;
   double                     d;
   clock_t                    t1;
   static clock_t             t0 = 0;
 
   if( t0 == 0 ) t0 = clock();
   t1 = clock() - t0;
   d = (double)(t1) / cps;
   return( d );
}
double RTSEC_SAME()
{
   struct timeval             tp;
   static long                start=0, startu;

   if( !start )
   {
      (void) gettimeofday( &tp, NULL );
      start  = tp.tv_sec;
      startu = tp.tv_usec;
      return( HPL_rzero );
   }
   (void) gettimeofday( &tp, NULL );

   return( (double)( tp.tv_sec - start ) +
           ( (double)( tp.tv_usec-startu ) / 1000000.0 ) );
}

u64Int
HPCC_starts_SAME(s64Int n)
{
  int i, j;
  u64Int m2[64];
  u64Int temp, ran;

  while (n < 0) n += PERIOD;
  while (n > PERIOD) n -= PERIOD;
  if (n == 0) return 0x1;

  temp = 0x1;
  for (i=0; i<64; i++) {
    m2[i] = temp;
    temp = (temp << 1) ^ ((s64Int) temp < 0 ? POLY : 0);
    temp = (temp << 1) ^ ((s64Int) temp < 0 ? POLY : 0);
  }

  for (i=62; i>=0; i--)
    if ((n >> i) & 1)
      break;

  ran = 0x2;
  while (i > 0) {
    temp = 0;
    for (j=0; j<64; j++)
      if ((ran >> j) & 1)
        temp ^= m2[j];
    ran = temp;
    i -= 1;
    if ((n >> i) & 1)
      ran = (ran << 1) ^ ((s64Int) ran < 0 ? POLY : 0);
  }

  return ran;
}

static __global__ void
d_init(size_t n, uint64_t *t)
{
  for (ptrdiff_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
    i += gridDim.x * blockDim.x) { 
    t[i] = i;
    }
}

static __device__ uint64_t
d_starts(size_t n)
{
  if (n == 0) {
    return 1;
  }

  int i = 63 - __clzll(n);

  uint64_t ran = 2;
  while (i > 0) {
    uint64_t temp = 0;
    for (int j = 0; j < 64; j++) {
      if ((ran >> j) & 1) {
        temp ^= c_m2[j];
      }
    }
    ran = temp;
    i -= 1;
    if ((n >> i) & 1) {
      ran = (ran << 1) ^ ((int64_t) ran < 0 ? POLY : 0);
    }
  }

  return ran;
}

__global__ void
d_bench(size_t n, uint64_t *t)
{
  size_t num_threads = gridDim.x * blockDim.x;
  size_t thread_num = blockIdx.x * blockDim.x + threadIdx.x;
  size_t start = thread_num * 4 * n / num_threads;
  size_t end = (thread_num + 1) * 4 * n / num_threads;
  uint64_t ran;
  ran = d_starts(start);
  for (ptrdiff_t i = thread_num; i < n; i += num_threads ) {
  //for (ptrdiff_t i = start; i < end; ++i) {
    ran = (ran << 1) ^ ((int64_t) ran < 0 ? POLY : 0);
    unsigned long long int *address, old, assumed;
    address = (unsigned long long int *)&t[ran & (n - 1)];
    old = *address;   
    do {
      assumed = old;
      old = atomicCAS(address, assumed, assumed ^ ran);
    } while  (assumed != old);
  }
}

static __global__ void
d_check(size_t TableSize, uint64_t *t)
{
  for (ptrdiff_t i = blockIdx.x * blockDim.x + threadIdx.x; i < TableSize;
       i += gridDim.x * blockDim.x) {
    if (t[i] != i) {
      atomicAdd(d_error, 1);
    }
  }
}
static void
starts()
{
  uint64_t m2[64];
  uint64_t temp = 1;
  for (ptrdiff_t i = 0; i < 64; i++) {
    m2[i] = temp;
    temp = (temp << 1) ^ ((int64_t) temp < 0 ? POLY : 0); 
    temp = (temp << 1) ^ ((int64_t) temp < 0 ? POLY : 0); 
  }
  hipMemcpyToSymbol(HIP_SYMBOL(c_m2), m2, sizeof(m2));
}


static void
RandomAccessUpdate(u64Int TableSize, u64Int *Table) {
  

  //d_init<<<grid, thread>>>(TableSize, d_t);
  //d_bench<<<grid, thread>>>(TableSize, d_t);



}
extern "C"
int
HPCC_RandomAccess(HPCC_Params *params, int doIO, double *GUPs, int *failure) {
  u64Int i;
  u64Int temp;
  double cputime;               /* CPU time to update table */
  double realtime;              /* Real time to update table */
  double totalMem;
  u64Int *Table;
  u64Int logTableSize, TableSize;
  FILE *outFile = NULL;

  if (doIO) {
    outFile = fopen( params->outFname, "a" );
    if (! outFile) {
      outFile = stderr;
      fprintf( outFile, "Cannot open output file.\n" );
      return 1;
    }
  }

  /* calculate local memory per node for the update table */
  totalMem = params->HPLMaxProcMem;
  totalMem /= sizeof(u64Int);

  /* calculate the size of update array (must be a power of 2) */
  for (totalMem *= 0.5, logTableSize = 0, TableSize = 1;
       totalMem >= 1.0;
       totalMem *= 0.5, logTableSize++, TableSize <<= 1)
    ; /* EMPTY */

  Table = HPCC_XMALLOC( u64Int, TableSize );
  if (! Table) {
    if (doIO) {
      fprintf( outFile, "Failed to allocate memory for the update table (" FSTR64 ").\n", TableSize);
      fclose( outFile );
    }
    return 1;
  }
  params->RandomAccess_N = (s64Int)TableSize;

  /* Print parameters for run */
  if (doIO) {
  fprintf( outFile, "Main table size   = 2^" FSTR64 " = " FSTR64 " words\n", logTableSize,TableSize);
  fprintf( outFile, "Number of updates = " FSTR64 "\n", NUPDATE);
  }

  /* Initialize main table */
  for (i=0; i<TableSize; i++) Table[i] = i;

  /* Initialize gpu */
  
  starts();

  int ndev;
  hipGetDeviceCount(&ndev);
  int dev = 0;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, dev);
  hipSetDevice(dev);
  uint64_t *d_t;
  hipMalloc((void **)&d_t, TableSize * sizeof(uint64_t)) ;

  dim3 grid(prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / prop.warpSize));
  dim3 thread(prop.warpSize);

  d_init<<<grid, thread>>>(TableSize, d_t);


  /* Begin timing here */
  cputime = -CPUSEC_SAME();
  realtime = -RTSEC_SAME();

  d_bench<<<grid, thread>>>(TableSize, d_t); // core
  hipMemcpy(Table, d_t, sizeof(uint64_t) * TableSize, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();//@lq
  //RandomAccessUpdate( TableSize, Table );

  /* End timed section */
  cputime += CPUSEC_SAME();
  realtime += RTSEC_SAME();

  /* make sure no division by zero */
  *GUPs = (realtime > 0.0 ? 1.0 / realtime : -1.0);
  *GUPs *= 1e-9*NUPDATE;
  /* Print timing results */
  if (doIO) {
  fprintf( outFile, "CPU time used  = %.6f seconds\n", cputime);
  fprintf( outFile, "Real time used = %.6f seconds\n", realtime);
  fprintf( outFile, "%.9f Billion(10^9) Updates    per second [GUP/s]\n", *GUPs );
  }

  /* Verification of results (in serial or "safe" mode; optional) */
  temp = 0x1;
  for (i=0; i<NUPDATE; i++) {
    temp = (temp << 1) ^ (((s64Int) temp < 0) ? POLY : 0);
    Table[temp & (TableSize-1)] ^= temp;
  }

  temp = 0;
  for (i=0; i<TableSize; i++)
    if (Table[i] != i)
      temp++;

  if (doIO) {
  fprintf( outFile, "Found " FSTR64 " errors in " FSTR64 " locations (%s).\n",
           temp, TableSize, (temp <= 0.01*TableSize) ? "passed" : "failed");
  }
  if (temp <= 0.01*TableSize) *failure = 0;
  else *failure = 1;

  HPCC_free( Table );

  if (doIO) {
    fflush( outFile );
    fclose( outFile );
  }




   //debug
  d_bench<<<grid, thread>>>(TableSize, d_t);
  void *p_error;
  hipGetSymbolAddress(&p_error, HIP_SYMBOL(d_error));
  hipMemset(d_error, 0, sizeof(uint32_t));
  d_check<<<grid, thread>>>(TableSize, d_t);
  uint32_t h_error;
  hipMemcpy(&h_error, p_error, sizeof(uint32_t), hipMemcpyDeviceToHost);
  printf("Verification: Found %u errors.\n", h_error);
  //

  hipFree(d_t);

  return 0;
}

